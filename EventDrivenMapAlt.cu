#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <ctime>
#include <cstdlib>
#include <armadillo>
#include <hiprand.h>
#include <assert.h>
#include "EventDrivenMap.hpp"
#include "parameters.hpp"

// Specialised constructor
EventDrivenMap::EventDrivenMap(const arma::vec* pParameters, unsigned int noReal)
{

  // Read in parameters
  mpHost_p = new arma::fvec((*pParameters).n_elem);
  *mpHost_p = arma::conv_to<arma::fvec>::from(*pParameters);

  // TEST
  printf("Beta = %f\n",(*mpHost_p)[0]);

  // Declare memory for temporary storage of U and F
  mpU = new arma::fvec(noSpikes);
  mpF = new arma::fvec(noSpikes);

  // CUDA stuff
  mNoReal    = noReal;
  mNoThreads = noThreads;
  mNoBlocks  = (mNoReal+mNoThreads-1)/mNoThreads;

  // Set initial time horizon
  mFinalTime = timeHorizon;

  // allocate memory on CPU
  mpHost_lastSpikeInd = (unsigned short*) malloc( noSpikes*sizeof(short));

  // allocate memory on GPU
  hipMalloc( &mpDev_p, mpHost_p->n_elem*sizeof(float) );
  hipMalloc( &mpDev_beta, mNoReal*mNoThreads*sizeof(float) );
  hipMalloc( &mpDev_v, mNoReal*mNoThreads*sizeof(float) );
  hipMalloc( &mpDev_s, mNoReal*mNoThreads*sizeof(float) );
  hipMalloc( &mpDev_w, mNoThreads*sizeof(float) );
  hipMalloc( &mpDev_U, (noSpikes+1)*sizeof(float) );
  hipMalloc( &mpDev_lastSpikeInd, mNoReal*mNoSpikes*sizeof(unsigned short) );
  hipMalloc( &mpDev_lastSpikeTime, mNoReal*mNoSpikes*sizeof(float) );
  hipMalloc( &mpDev_crossedSpikeInd, mNoReal*mNoSpikes*sizeof(unsigned short) );
  hipMalloc( &mpDev_crossedSpikeTime, mNoReal*mNoSpikes*sizeof(float) );

  // Set up coupling kernel
  BuildCouplingKernel();

  // Copy parameters over
  hipMemcpy(mpDev_p,mpHost_p->begin(),(*mpHost_p).n_elem*sizeof(float),hipMemcpyHostToDevice);

  // initialise random number generators
  hiprandCreateGenerator( &mGen, HIPRAND_RNG_PSEUDO_DEFAULT);
  ResetSeed();
  mParStdDev = 0.0f;

  // For testing
  mpHostData = (float*) malloc( noReal*mNoStoredSpikes*sizeof(float));
}

void EventDrivenMap::BuildCouplingKernel()
{
  float *w;
  w = (float*) malloc( mNoThreads*sizeof(float));
  for (int i=0;i<mNoThreads;++i)
  {
    float x = -L + (float)(2*L/mNoThreads)*i;
    w[i] = (a1*exp(-b1*abs(x))-a2*exp(-b2*abs(x)))*2*L/mNoThreads;
  }
  circshift(w,mNoThreads/2);
  hipMemcpy(mpDev_w,w,mNoThreads*sizeof(float),hipMemcpyHostToDevice);
  FILE *fp = fopen("test.dat","w");
  for (int i=0;i<mNoThreads;++i)
  {
    fprintf(fp,"%f\n",w[i]);
  }
  fclose(fp);
  free(w);
}

EventDrivenMap::~EventDrivenMap()
{
  delete mpU;
  delete mpF;
  delete mpHost_p;

  free(mpHost_lastSpikeInd);

  hipFree(mpDev_p);
  hipFree(mpDev_beta);
  hipFree(mpDev_v);
  hipFree(mpDev_s);
  hipFree(mpDev_w);
  hipFree(mpDev_U);
  hipFree(mpDev_lastSpikeInd);
  hipFree(mpDev_lastSpikeTime);
  hipFree(mpDev_crossedSpikeInd);
  hipFree(mpDev_crossedSpikeTime);

  hiprandDestroyGenerator(mGen);

  free(mpHostData);
}

void EventDrivenMap::ComputeF(const arma::vec& Z, arma::vec& f)
{

  arma::vec U0(noSpikes+1);
  arma::vec UT(noSpikes);
  arma::fvec fU(noSpikes+1);

  // First, put vector in correct form
  ZtoU(Z,U0);

  // Then, typecast data as floats
  fU = arma::conv_to<arma::fvec>::from(U0);

  // Assuming that weight kernel does not change
  hipMemcpy(mpDev_U,fU.begin(),(noSpikes+1)*sizeof(float),hipMemcpyHostToDevice);

  // Introduce parameters heterogeneity
  hiprandGenerateNormal( mGen, mpDev_beta, mNoReal*mNoThreads, (*mpHost_p)[0], mParStdDev);

  // Lift - working
  LiftKernel<<<mNoReal,mNoThreads>>>(mpDev_s,mpDev_v,mpDev_p,mpDev_U,mNoReal);
  float *mpHost_Lift;
  mpHost_Lift = (float*) malloc( 2*mNoThreads*sizeof(float));
  hipMemcpy( mpHost_Lift,mpDev_v,mNoThreads*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy( mpHost_Lift+mNoThreads,mpDev_s,mNoThreads*sizeof(float),hipMemcpyDeviceToHost);

  FILE *fp = fopen("testLift.dat","w");
  for (int i=0;i<mNoThreads;++i)
  {
    fprintf(fp,"%f\t%f\n",mpHost_Lift[i],mpHost_Lift[i+mNoThreads]);
  }
  fclose(fp);
  free(mpHost_Lift);

  // Copy data to GPU
  //hipMemcpy( dev_w, w, mNoThreads*sizeof(float), hipMemcpyHostToDevice );
  // Save spike indices
  initialSpikeInd( U0);
  hipMemcpy( mpDev_lastSpikeInd, mpHost_lastSpikeInd, noSpikes*sizeof(unsigned short), hipMemcpyHostToDevice );

  // Evolve - working
  printf("Time horizon = %f\n",mFinalTime);
  float par_val;
  hipMemcpy(&par_val,mpDev_p,sizeof(float),hipMemcpyDeviceToHost);
  printf("Beta = %f\n",par_val);
  EvolveKernel<<<mNoReal,mNoThreads>>>(mpDev_v,mpDev_s,mpDev_p,mpDev_w,mFinalTime,mpDev_lastSpikeInd,mpDev_lastSpikeTime,
      mpDev_crossedSpikeInd,mpDev_crossedSpikeTime);

  // Restrict
  RestrictKernel<<<noSpikes*mNoReal,mNoThreads>>>( mpDev_lastSpikeTime, mpDev_lastSpikeInd,
      mpDev_crossedSpikeTime, mpDev_crossedSpikeInd, mFinalTime);

  //char filename3[] = "testAverages.dat";
  //float *mpHost_averages;
  //mpHost_averages = (float*) malloc( 5*noSpikes*mNoReal*sizeof(float));
  //hipMemcpy( mpHost_averages, mpDev_averages, 5*noSpikes*mNoReal*sizeof(float), hipMemcpyDeviceToHost);
  //SaveData(5*noSpikes*mNoReal,mpHost_averages,filename3);
  //free(mpHost_averages);

  realisationReductionKernelBlocks<<<noSpikes,mNoThreads>>>(
      mpDev_U, mpDev_lastSpikeTime, mNoReal);

  printf("Averaging done.\n");

  // Copy data back to CPU
  fU.resize(noSpikes);
  hipMemcpy( fU.begin(), mpDev_U, noSpikes*sizeof(float), hipMemcpyDeviceToHost );

  printf("U = \n");
  std::cout << fU << std::endl;

  // Compute F
  UT = arma::conv_to<arma::vec>::from(fU);

  f = U0[0]*U0.rows(1:noSpikes) - UT + U[0]*mFinalTime;
}

void EventDrivenMap::SetTimeHorizon( const float T)
{
  assert(T>0);
  mFinalTime = T;
}

void EventDrivenMap::SetNoRealisations( const int noReal)
{
  assert(noReal>0);
  mNoReal = noReal;
}

void EventDrivenMap::SetParameterStdDev( const float sigma)
{
  assert(sigma>=0);
  mParStdDev = sigma;
}

void EventDrivenMap::SetParameters( const unsigned int parId, const float parVal)
{
  assert(parId>=0);
  assert(parId<=(*mpHost).n_elem);
  (*mpHost_p)[parId] = parVal;
  hipMemcpy(mpDev_p+parId,&parVal,sizeof(float),hipMemcpyHostToDevice);
}

void EventDrivenMap::SetStorageCapacity( const unsigned int storageCapacity)
{
  assert(storageCapacity>0);
  mNoStoredSpikes = storageCapacity;
  realloc( mpDev_firingInd, mNoReal*mNoStoredSpikes*sizeof(short));
  realloc( mpDev_firingTime, mNoReal*mNoStoredSpikes*sizeof(float));
  realloc( mpDev_spikeInd, mNoReal*mNoStoredSpikes*sizeof(short));
}

void EventDrivenMap::ResetSeed()
{
  hiprandSetPseudoRandomGeneratorSeed( mGen, (unsigned long long) clock() );
}

void EventDrivenMap::initialSpikeInd( const arma::vec& U)
{
  unsigned int i,m;
  mpHost_lastSpikeInd[0] = mNoThreads/2;
  for (m=1;m<noSpikes;m++) {
    for (i=mpHost_lastSpikeInd[m-1];i>0;i--) {
      if (-L+(float)(2*i*L/mNoThreads)<-U[0]*U[m]) {
        mpHost_lastSpikeInd[m] = i;
        break;
      }
    }
  }
}

void EventDrivenMap::ZtoU( const arma::vec& Z, arma::vec& U)
{
  assert(U.n_elem==Z.n_elem+1);
  U[0] = Z[0];
  U[1] = 0.0;
  for (int i=2;i<=noSpikes;i++) {
    U[i] = Z[i-1];
  }
}

void EventDrivenMap::UtoZ( const arma::vec *U, arma::vec *Z)
{
  Z[0] = U[0];
  for (int i=1;i<noSpikes;i++) {
    Z[i] = U[i+1];
  }
}

__global__ void LiftKernel( float *S, float *v, const float *par, const float *U,
    const unsigned int noReal)
{
  int k = threadIdx.x + blockIdx.x*blockDim.x;
  int m;
  if(k<noThreads*noReal){

    //Define x-array
    float x = L - (float)(2*L/noThreads)*threadIdx.x;
    float s = 0.0f;
    float c = U[0];
    float beta = par[0];
    float dummyV, dummyS = 0.0f;

    // Lift Voltage
    # pragma unroll
    for(m=1; m<=noSpikes;m++){
      dummyV = ((x-c*U[m]>0.0f)*(((a1*beta*c)/((beta+c*b1)*(1.0f+c*b1)))* exp(c*U[m]*((1.0f+c*b1)/c))*exp(-b1*c*U[m])
              - ((a2*beta*c)/((beta+c*b2)*(1.0f+c*b2)))* exp(c*U[m]*((1.0f+c*b2)/c))*exp(-b2*c*U[m])+(a1*beta*c/(1.0f-beta))*exp(beta*U[m])*(1.0f/(beta+c*b1)+ 1.0f/(c*b1 - beta))*(exp((x/c)*(1.0f-beta))-exp(((c*U[m])/c)*(1.0f-beta)))-(a1*beta*c/((-beta+c*b1)*(1.0f-c*b1)))*exp(b1*c*U[m])*(exp(x*((1.0f-c*b1)/c))-exp(c*U[m]*((1.0f-c*b1)/c)))
               -(a2*beta*c/(1.0f-beta))*exp(beta*U[m])*(1.0f/(beta+c*b2) + 1.0f/(c*b2 - beta))*(exp((x/c)*(1.0f-beta))-exp((U[m])*(1.0f-beta)))
              +(a2*beta*c/((-beta+c*b2)*(1.0f-c*b2)))*exp(b2*c*U[m])*(exp(x*((1.0f-c*b2)/c))-exp(c*U[m]*((1.0f-c*b2)/c))))
               +
            (x-c*U[m]<=0.0f)*(((a1*beta*c)/((beta +c*b1)*(1.0f+c*b1)))*(exp(x*((1.0f+c*b1)/c)))*exp(-b1*c*U[m])
               - ((a2*beta*c)/((beta +c*b2)*(1.0f+c*b2)))*(exp(x*((1.0f+c*b2)/c)))*exp(-b2*c*U[m])))*exp(-x/c);

      s += dummyV - ((x - c*U[m])>0.0f)*exp(-(x-c*U[m])/c) + ((x-c*U[m])<=0.0f)*0.0f;

      dummyS += ((c*U[m]-x)>0.0f)*(beta*a1*(c/(beta +c*b1))*exp(b1*(x- c*U[m])) - beta*a2*(c/(beta+c*b2))*exp(b2*(x- c*U[m])))
        +((c*U[m]-x)<= 0.0f)*((2.0f*a1/b1)*(beta/(1.0f - ((beta*beta)/(c*c*b1*b1))))*exp(-(beta/c)*(x-c*U[m])) -beta*a1*(c/(-beta +c*b1))*(exp(b1*(c*U[m] - x)))
        - (2.0f*a2/b2)*(beta/(1.0f - ((beta*beta)/(c*c*b2*b2))))*exp(-(beta/c)*(x-c*U[m])) + beta*a2*(c/(-beta +c*b2))*(exp(b2*(c*U[m] - x))));
    }

    v[k] = I + s;
    v[k] *= (v[k]<1.0f);
    S[k] = dummyS;
  }

}

__device__ float fun( float t, float v, float s, float beta)
{
  return v*exp(-t)+I*(1.0f-exp(-t))+s*exp(-t)/(1.0f-beta)*(exp((1.0f-beta)*t)-1.0f)-vth;
}

__device__ float dfun( float t, float v, float s, float beta)
{
  return I*exp(-t)-v*exp(-t)+s*exp(-t)*exp(-t*(beta-1))+(s*exp(-t)*(exp(-t*(beta-1))-1.0f))/(beta-1);
}

__device__ float eventTime( float v0, float s0, float beta)
{
  int decision;
  float f, df, estimatedTime = 0.0f;
  decision = (int) (v0>vth*pow(s0/(vth-I),1.0f/beta)+I*(1.0f-pow(s0/(vth-I),1.0f/beta))-(vth-I)/(beta-1.0f)*(s0/(vth-I)-pow(s0/(vth-I),1.0f/beta)));

  f  = fun( estimatedTime, v0, s0, beta)*decision;
  df = dfun( estimatedTime, v0, s0, beta);

  while (abs(f)>tol) {
    estimatedTime -= f/df;
    f  = fun( estimatedTime, v0, s0, beta);
    df = dfun( estimatedTime, v0, s0, beta);
  }

  return estimatedTime+100.0f*(1.0f-decision);

}

__global__ void EvolveKernel( float *v, float *s, const float *beta,
    const float *w, const float finalTime, unsigned short *global_lastSpikeInd,
    float *global_lastSpikeTime, unsigned short *global_crossedSpikeInd,
    float *global_crossedSpikeTime)
{
  __shared__ unsigned short local_lastSpikeInd[noSpikes];
  __shared__ unsigned short local_crossedSpikeInd[noSpikes];
  __shared__ float local_lastSpikeTime[noSpikes];
  __shared__ float local_crossedSpikeTime[noSpikes];
  float currentTime = 0.0f;
  float local_v, local_s, local_beta;
  unsigned int crossings;
  unsigned short minIndex;
  struct EventDrivenMap::firing val;

  // load values from global memory
  local_v = v[threadIdx.x+blockIdx.x*blockDim.x];
  local_s = s[threadIdx.x+blockIdx.x*blockDim.x];
  local_beta = beta[threadIdx.x+blockIdx.x*blockDim.x];

  if (threadIdx.x<noSpikes) {
    local_lastSpikeInd[threadIdx.x] = lastSpikeInd[threadIdx.x];
  }
  while (crossings<(2<<noSpikes)-1)
  {
    // find next firing times
    val.time  = eventTime(local_v,local_s,local_beta);
    val.index = threadIdx.x;

    // perform reduction to find minimum spike time
    val = blockReduceMin( val);

    // val now contains minimum spike time and index
    // update values to spike time
    local_v *= exp(-val.time);
    local_v +=
      I*(1.0f-exp(-val.time))+local_s*exp(-val.time)/(1.0f-local_beta)*(exp((1.0f-local_beta)*val.time)-1.0f);
    local_v *= (threadIdx.x!=val.index);
    local_s *= exp(-local_beta*val.time);
    local_s += local_beta*w[(threadIdx.x-val.index)*(threadIdx.x>=val.index)+(val.index-threadIdx.x)*(threadIdx.x<val.index)];

    currentTime += val.time;

    // store values
    if (threadIdx.x==0)
    {
      // First calculate which crossing spike belongs to
      minIndex = 0;
      for (int i=1;i<noSpikes;i++)
      {
        minIndex += ((std::abs((int)(val.index-local_lastSpikeInd[i])))<(std::abs((int)(val.index-local_lastSpikeInd[minIndex]))));
      }
      if (crossings & (1<<minIndex-1))
      {
        if (currentTime>finalTime)
        {
          local_crossedSpikeTime[minIndex] = currentTime;
          local_crossedSpikeInd[minIndex]  = val.index;
          crossings += (1<<minIndex-1);
        }
        else
        {
          local_lastSpikeTime[minIndex] = currentTime;
          local_lastSpikeInd[minIndex]  = val.index;
        }
      }
    }

    // Save into global memory
    if (threadIdx.x<noSpikes)
    {
      global_lastSpikeInd[blockIdx.x*noSpikes+threadIdx.x]     = local_lastSpikeInd[threadIdx.x];
      global_lastSpikeTime[blockIdx.x*noSpikes+threadIdx.x]    = local_lastSpikeTime[threadIdx.x];
      global_crossedSpikeInd[blockIdx.x*noSpikes+threadIdx.x]  = local_crossedSpikeInd[threadIdx.x];
      global_crossedSpikeTime[blockIdx.x*noSpikes+threadIdx.x] = local_crossedSpikeTime[threadIdx.x];
    }
  }
}

/* Restrict functions */
__global__ void RestrictKernel( float *global_lastSpikeTime,
                                const unsigned short *global_lastSpikeInd,
                                const float *global_crossedSpikeTime
                                const unsigned short *global_crossedSpikeInd,
                                const float finalTime)
{
  unsigned int index = threadIdx.x+blockDim.x*blockIdx.x;
  float t0 = global_lastSpikeTime[index];
  float t1 = global_crossedSpikeTime[index];
  unsigned short x0 = -L+2.0f*L/NoThreads*global_lastSpikeInd[index];
  unsigned short x1 = -L+2.0f*L/NoThreads*global_crossedSpikeInd[index];
  global_lastSpikeTime[index] = x0+(finalTime-t0)*(x1-x0)/(t1-t0);
}

__global__ void realisationReductionKernelBlocks( float *V,
                                                  const float *U,
                                                  const unsigned int noReal)
{
  unsigned int i, spikeNo = blockIdx.x;
  unsigned int index;
  unsigned int noLoad = (noReal+blockDim.x-1)/blockDim.x;
  float average = 0.0f;

  for (i=0;i<noLoad;i++) {
    index = threadIdx.x+i*blockDim.x;
    average += (index < noReal) ? U[index+spikeNo*noReal] : 0.0f;
  }
  average = blockReduceSum( average);
  if (threadIdx.x==0) {
    V[spikeNo] = average/noReal;
  }
}

void circshift( float *w, int shift) {
  int i;
  float dummy[noThreads];
  # pragma unroll
  for (i=0;i<noThreads-shift;i++) {
    dummy[i] = w[shift+i];
  }
  # pragma unroll
  for (i=0;i<shift;i++) {
    dummy[noThreads-shift+i] = w[i];
  }
  # pragma unroll
  for (i=0;i<noThreads;i++) {
    w[i] = dummy[i];
  }
}

__device__ struct EventDrivenMap::firing warpReduceMin( struct EventDrivenMap::firing val) {
  float dummyTime;
  unsigned int dummyIndex;
  for (int offset = warpSize/2; offset>0; offset/=2) {
    dummyTime  = __shfl_down( val.time, offset);
    dummyIndex = __shfl_down( val.index, offset);
    val.time   = (val.time < dummyTime) ? val.time : dummyTime;
    val.index  = (val.time < dummyTime) ? val.index : dummyIndex;
  }
  return val;
}

__device__ struct EventDrivenMap::firing blockReduceMin( struct EventDrivenMap::firing val) {
  __shared__ struct EventDrivenMap::firing shared[32];
  int lane = threadIdx.x % warpSize;
  int wid  = threadIdx.x / warpSize;

  val = warpReduceMin( val);

  if (lane==0) {
    shared[wid] = val;
  }
  __syncthreads();

  val.time  = (threadIdx.x<blockDim.x/warpSize) ? shared[lane].time  : 0.0f;
  val.index = (threadIdx.x<blockDim.x/warpSize) ? shared[lane].index : 0;

  if (wid==0) {
    val = warpReduceMin( val);
  }

  if (threadIdx.x==0) {
    shared[0] = val;
  }
  __syncthreads();
  val = shared[0];

  return val;
}

__device__ float warpReduceSum( float val) {
  for (int offset = warpSize/2; offset>0; offset/=2) {
    val += __shfl_down( val, offset);
  }
  return val;
}

__device__ float blockReduceSum( float val) {
  __shared__ float shared[32];
  int lane = threadIdx.x % warpSize;
  int wid  = threadIdx.x / warpSize;

  val = warpReduceSum( val);

  if (lane==0) {
    shared[wid] = val;
  }
  __syncthreads();

  val = (threadIdx.x<blockDim.x/warpSize) ? shared[lane] : 0.0f;

  if (wid==0) {
    val = warpReduceSum( val);
  }

  return val;
}

/* These functions are to help with doing reductions */
__device__ struct EventDrivenMap::averaging warpReduceSumSimultaneous( struct
    EventDrivenMap::averaging val)
{
  for (int offset = warpSize/2; offset>0; offset/=2) {
    val.t   += __shfl_down( val.t, offset);
    val.x   += __shfl_down( val.x, offset);
    val.tSq += __shfl_down( val.tSq, offset);
    val.xt  += __shfl_down( val.xt, offset);
    val.count += __shfl_down( val.count, offset);
  }
  return val;
}

__device__ struct EventDrivenMap::averaging blockReduceSumSimultaneous( struct
    EventDrivenMap::averaging val)
{
  __shared__ struct EventDrivenMap::averaging shared[32];
  int lane = threadIdx.x % warpSize;
  int wid  = threadIdx.x / warpSize;

  val = warpReduceSumSimultaneous( val);

  if (lane==0) {
    shared[wid] = val;
  }
  __syncthreads();

  val.t     = (threadIdx.x<blockDim.x/warpSize) ? shared[lane].t : 0.0f;
  val.x     = (threadIdx.x<blockDim.x/warpSize) ? shared[lane].x : 0.0f;
  val.tSq   = (threadIdx.x<blockDim.x/warpSize) ? shared[lane].tSq : 0.0f;
  val.xt    = (threadIdx.x<blockDim.x/warpSize) ? shared[lane].xt : 0.0f;
  val.count = (threadIdx.x<blockDim.x/warpSize) ? shared[lane].count : 0;

  if (wid==0) {
    val = warpReduceSumSimultaneous( val);
  }

  return val;
}

void SaveData( int npts, float *x, char *filename) {
  FILE *fp = fopen(filename,"w");
  for (int i=0;i<npts;i++) {
    fprintf(fp,"%f\n",x[i]);
  }
  fclose(fp);
}
